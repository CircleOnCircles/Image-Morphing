#include "hip/hip_runtime.h"
#include <cfloat>

__constant__ KernParameters c_params;

const int OPT_BW = 32,
          OPT_BH = 8,
          OPT_NB = 5;

const int SPACING = 5;

template <int BW, int BH>
struct SSIMData
{
    float2 mean[BH*2+4][BW*2+4],
           var[BH*2+4][BW*2+4];
    float  cross[BH*2+4][BW*2+4],
           value[BH*2+4][BW*2+4];

    int2 orig;
};

template <class T>
__device__ void swap(T &a, T &b)/*{{{*/
{
    T temp = a;
    a = b;
    b = temp;
}/*}}}*/

// returns -1 if pixel cannot improve due to neighbors (and itself) 
// not improving
__device__ int get_improve_mask_idx(const KernPyramidLevel &lvl, /*{{{*/
                            const int2 &p)
{
    int2 block = p/5;
    int2 offset = p%5;

    int begi = (offset.y >= 2 ? 1 : 0),
        begj = (offset.x >= 2 ? 1 : 0),
        endi = begi+2,
        endj = begj+2;

    int impmask_idx = (block.y+1)*lvl.impmask_rowstride + (block.x+1);

    for(int i=begi; i<endi; ++i)
    {
        for(int j=begj; j<endj; ++j)
        {
            int d = impmask_idx + c_improvmask_offset[i][j];

            if(lvl.improving_mask[d]&c_improvmask[offset.y][offset.x][i][j])
                return impmask_idx;
        }
    }

    return -1;
}/*}}}*/

__device__ bool pixel_on_border(const KernPyramidLevel &lvl, const int2 &p)/*{{{*/
{
    switch(c_params.bcond)
    {
    case BCOND_NONE:
        break;
    case BCOND_CORNER:
        if(p.x==0 && p.y==0 || p.x==0 && p.y==lvl.pixdim.y-1 ||
           p.x==lvl.pixdim.x-1 && p.y==0 && p.x==lvl.pixdim.x-1 && p.y==lvl.pixdim.y-1)
        {
            return true;
        }
        break;
    case BCOND_BORDER:
        if(p.x==0 || p.y==0 || p.x==lvl.pixdim.x-1 || p.y==lvl.pixdim.y-1)
            return true;
        break;
    }
    return false;
}/*}}}*/

// gradient calculation --------------------------

template <int BW, int BH>
__device__ float ssim_change(const KernPyramidLevel &lvl,/*{{{*/
                            const int2 &p,
                            float2 v, float2 old_luma, 
                            const SSIMData<BW,BH> &ssimdata)
{
    float2 luma;

    luma.x = tex2D(tex_img0, p.x-v.x + 0.5f, p.y-v.y + 0.5f),
    luma.y = tex2D(tex_img1, p.x+v.x + 0.5f, p.y+v.y + 0.5f);

    float change = 0;

    float2 dmean = luma - old_luma,
           dvar = pow2(luma) - pow2(old_luma);
    float  dcross = luma.x*luma.y - old_luma.x*old_luma.y;

    bool need_counter = p.x < 4 || p.x >= lvl.pixdim.x-4 ||
                        p.y < 4 || p.y >= lvl.pixdim.y-4;

    int idx = mem_index(lvl, p);
    int2 B = calc_border(p, lvl.pixdim);

    for(int i=0; i<5; ++i)
    {
        int sy = p.y+i-2 - ssimdata.orig.y;
        assert(sy >= 0 && sy < OPT_BH*2+4);
        for(int j=0; j<5; ++j)
        {
            if(c_iomask[B.y][B.x][i][j] == 0)
                continue;

            int sx = p.x+j-2 - ssimdata.orig.x;

            int nb = mem_index(lvl, p + make_int2(j,i)-2);

            float2 mean, var;
            float counter = need_counter ? lvl.ssim.counter[nb] : 25,
                  cross;

            assert(sx >= 0 && sx < OPT_BW*2+4);

            mean = ssimdata.mean[sy][sx];
            var = ssimdata.var[sy][sx];
            cross = ssimdata.cross[sy][sx];

            mean += dmean;
            var +=  dvar;
            cross += dcross;

            float new_ssim = ssim(mean,var,cross,counter,c_params.ssim_clamp);
            change += ssimdata.value[sy][sx] - new_ssim;
        }
    }

    return change;
}/*}}}*/

template <int BW, int BH>
__device__ float energy_change(const KernPyramidLevel &lvl, /*{{{*/
                               const int2 &p,
                               const float2 &v,
                               const float2 &old_luma,
                               const float2 &d,
                               const SSIMData<BW,BH> &ssimdata)
{
    float v_ssim = ssim_change(lvl, p, v+d, old_luma, ssimdata);

    int idx = mem_index(lvl,p);

    float v_tps = lvl.tps.axy[idx]*(d.x*d.x + d.y*d.y);
    v_tps += lvl.tps.b[idx].x*d.x;
    v_tps += lvl.tps.b[idx].y*d.y;

    float v_ui  = lvl.ui.axy[idx]*(d.x*d.x + d.y*d.y);
    v_ui += lvl.ui.b[idx].x*d.x;
    v_ui += lvl.ui.b[idx].y*d.y;

    return (c_params.w_ui*v_ui + c_params.w_ssim*v_ssim)*lvl.inv_wh
                + c_params.w_tps*v_tps;
}/*}}}*/

template <int BW, int BH>
__device__ float2 compute_gradient(const KernPyramidLevel &lvl, /*{{{*/
                                   const int2 &p,
                                   const float2 &v,
                                   const float2 &old_luma,
                                   const SSIMData<BW,BH> &ssimdata)
{
    float2 g;
    g.x = energy_change(lvl,p,v,old_luma,make_float2(c_params.eps,0),ssimdata)-
          energy_change(lvl,p,v,old_luma,make_float2(-c_params.eps,0),ssimdata);
    g.y = energy_change(lvl,p,v,old_luma,make_float2(0,c_params.eps),ssimdata)-
          energy_change(lvl,p,v,old_luma,make_float2(0,-c_params.eps),ssimdata);
    return -g;
}/*}}}*/

// foldover --------------------------------

template <int X, int Y, int SIGN>
__device__ float2 fover_calc_vtx(const KernPyramidLevel &lvl,/*{{{*/
                                 const int2 &p, float2 v)
{
    const int2 off = make_int2(X,Y);

    if(lvl.contains(p+off))
        v = SIGN*lvl.v[mem_index(lvl,p+off)];

     return v + (p-off);
}/*}}}*/

__device__ void fover_update_isec_min(float2 c, float2 grad,/*{{{*/
                                      float2 e0, float2 e1,
                                      float &t_min)
{
    float2 de = e1-e0,
           dce = c-e0;

    // determinant
    float d  = de.y*grad.x - de.x*grad.y;

    // signals that we don't have an intersection (yet)
    // t = td/d
    float td = -1;

    // u = ud/d
    // e0 + u*(e1-e0) = intersection point
    float ud = grad.x*dce.y - grad.y*dce.x;

    int sign = signbit(d);

    // this is faster than multiplying ud and d by sign
    if(sign)
    {
        ud = -ud;
        d = -d;
    }

    // line by c0 and c1 intersects segment [e0,e1] ?
    if(ud >= 0 && ud <= d) // u >= 0 && u <= 1
    {
        // c0 + t*(c1-c0) = intersection point
        td = de.x*dce.y - de.y*dce.x;
        td *= (-sign*2+1);

        if(td >= 0 && td < t_min*d)
            t_min = td/d;
    }
}/*}}}*/

template <int SIGN>
__device__ void fover_calc_isec_min(const KernPyramidLevel &lvl, /*{{{*/
                                    const int2 &p,
                                    float2 v, float2 grad, 
                                    float &t_min)
{
    // edge segment, start from upper left (-1,-1), go cw around center
    // pixel testing whether pixel will intersect the edge or not
    float2 e[2] = { fover_calc_vtx<-1,-1,SIGN>(lvl, p, v),
                    fover_calc_vtx< 0,-1,SIGN>(lvl, p, v)};

    float2 efirst = e[0];

    // pixel displacement (c0 -> c1)
    float2 c = p + v;

    fover_update_isec_min(c,grad,e[0],e[1],t_min);

    e[0]  = fover_calc_vtx<1,-1,SIGN>(lvl, p, v);
    fover_update_isec_min(c,grad,e[1],e[0],t_min);

    e[1]  = fover_calc_vtx<1,0,SIGN>(lvl, p, v);
    fover_update_isec_min(c,grad,e[0],e[1],t_min);

    e[0]  = fover_calc_vtx<1,1,SIGN>(lvl, p, v);
    fover_update_isec_min(c,grad,e[1],e[0],t_min);

    e[1]  = fover_calc_vtx<0,1,SIGN>(lvl, p, v);
    fover_update_isec_min(c,grad,e[0],e[1],t_min);

    e[0]  = fover_calc_vtx<-1,1,SIGN>(lvl, p, v);
    fover_update_isec_min(c,grad,e[1],e[0],t_min);

    e[1]  = fover_calc_vtx<-1,0,SIGN>(lvl, p, v);
    fover_update_isec_min(c,grad,e[0],e[1],t_min);

    fover_update_isec_min(c,grad,e[1],efirst,t_min);
}/*}}}*/

__device__ float prevent_foldover(const KernPyramidLevel &lvl,/*{{{*/
                                  const int2 &p, 
                                  float2 v, float2 grad)
{
    float t_min = 10;

    fover_calc_isec_min<-1>(lvl, p, -v, -grad, t_min);
    fover_calc_isec_min<1>(lvl, p, v, grad, t_min);

    return max(t_min-c_params.eps,0.0f);
}/*}}}*/

template <int BW, int BH>
__device__ void golden_section_search(const KernPyramidLevel &lvl,/*{{{*/
                                      const int2 &p,
                                      float a, float c,
                                      float2 v, float2 grad,
                                      float2 old_luma,
                                      const SSIMData<BW,BH> &ssimdata,
                                      float &fmin, float &tmin)
{
    const float R = 0.618033989f,
                C = 1.0f - R;


    float b = a*R + c*C,  // b between [a,c>
          x = b*R + c*C;  // x between [b,c>

    float fb = energy_change(lvl, p, v, old_luma, grad*b, ssimdata),
          fx = energy_change(lvl, p, v, old_luma, grad*x, ssimdata);

#pragma unroll 4
    while(c - a > c_params.eps)
    {
        if(fx < fb) // bracket is [b,x,c] ?
        {
            // [a,b,c] = [b,x,c]
            a = b;
            b = x;
            x = b*R + c*C; // x between [b,c>
        }
        else // bracket is [a,b,x] ?
        {
            // [a,b,c] = [a,b,x]
            c = x;
            x = b*R + a*C; // x between <a,b]
        }

        float f = energy_change(lvl, p, v, old_luma, grad*x, ssimdata);

        if(fx < fb)
        {
            fb = fx;
            fx = f;
        }
        else
        {
            swap(b,x);
            fx = fb;
            fb = f;
        }
    }

    if(fx < fb)
    {
        tmin = x;
        fmin = fx;
    }
    else
    {
        tmin = b;
        fmin = fb;
    }
}/*}}}*/

// update --------------------------------

template <int BW, int BH>
__device__ void ssim_update(KernPyramidLevel &lvl,/*{{{*/
                            const int2 &p, 
                            float2 v, float2 old_luma,
                            SSIMData<BW,BH> &ssimdata)
{
    float2 luma;

    luma.x = tex2D(tex_img0, p.x-v.x + 0.5f, p.y-v.y + 0.5f),
    luma.y = tex2D(tex_img1, p.x+v.x + 0.5f, p.y+v.y + 0.5f);

    int idx = mem_index(lvl,p);

    lvl.ssim.luma[idx] = luma;

    float2 dmean = luma - old_luma,
           dvar = pow2(luma) - pow2(old_luma);
    float  dcross = luma.x*luma.y - old_luma.x*old_luma.y;

    int2 B = calc_border(p, lvl.pixdim);

    for(int i=0; i<5; ++i)
    {
        int sy = p.y+i-2 - ssimdata.orig.y;
        for(int j=0; j<5; ++j)
        {
            if(c_iomask[B.y][B.x][i][j])
            {
                int sx = p.x+j-2 - ssimdata.orig.x;

                atomicAdd(&ssimdata.mean[sy][sx], dmean);
                atomicAdd(&ssimdata.var[sy][sx], dvar);
                atomicAdd(&ssimdata.cross[sy][sx], dcross);
            }
        }
    }
}/*}}}*/

template <int BW, int BH>
__device__ void commit_pixel_motion(KernPyramidLevel &lvl, /*{{{*/
                                    const int2 &p,
                                    const float2 &newv,
                                    const float2 &old_luma,
                                    const float2 &grad,
                                    SSIMData<BW,BH> &ssimdata)

{
    ssim_update(lvl, p, newv, old_luma, ssimdata);

    int2 B = calc_border(p, lvl.pixdim);

    // tps update
    for(int i=0; i<5; ++i)
    {
        for(int j=0; j<5; ++j)
        {
            assert(lvl.contains(p.x+j-2,p.y+i-2) || c_tps_data[B.y][B.x][i][j] == 0);

            int nb = mem_index(lvl, p + make_int2(j,i)-2);
            atomicAdd(&lvl.tps.b[nb], grad*c_tps_data[B.y][B.x][i][j]);
        }
    }

    int idx = mem_index(lvl,p);

    // ui update
    lvl.ui.b[idx] += 2*grad*lvl.ui.axy[idx];

    // vector update
    lvl.v[idx] = newv;
}/*}}}*/

// optimization kernel --------------------------

template <int BW, int BH>
__device__ bool optimize_pixel(const KernPyramidLevel &lvl,/*{{{*/
                               const int2 &p,
                               const SSIMData<BW,BH> &ssim,
                               float2 &old_luma,
                               float2 &v,
                               float2 &grad,
                               int &impmask_idx)
{
    if(lvl.contains(p))
    {
        int idx = mem_index(lvl,p);

        v = lvl.v[idx],
        old_luma = lvl.ssim.luma[idx];

        impmask_idx = get_improve_mask_idx(lvl, p);

        assert(lvl.contains(p) || lvl.improving_mask[impmask_idx] == 0);

        if(impmask_idx >= 0)
        {
            if(!pixel_on_border(lvl, p))
            {
                grad = compute_gradient(lvl, p, v, old_luma, ssim);

            //    float ng = hypot(grad.x,grad.y); // slower
                float ng = sqrt(pow2(grad.x)+pow2(grad.y));

                if(ng != 0)
                {
                    grad /= ng;

                    float t = prevent_foldover(lvl, p, v, grad);

                    float tmin, fmin;

                    golden_section_search(lvl, p, 0, t,
                                          v, grad, old_luma, ssim, fmin, tmin);

                    if(fmin < 0)
                    {
                        grad *= tmin;
                        v += grad;
                        return true;
                    }
                }
            }
        }
    }
    return false;
}/*}}}*/

template <template<int,int> class F>
__device__ void process_shared_state(F<8,8> fun, const KernPyramidLevel &lvl,/*{{{*/
                                     const int2 &block_orig)
{
    const int BW = 8, BH = 8;

    /*     BW      BW      4
       -----------------------
       |        |        |   | BH
       |   1    |   2    | 6 |
       |-----------------|---|
       |        |        |   | BH
       |   4    |   3    | 6 |
       |-----------------|---|
       |   5    |   5    | 7 | 4
       -----------------------
    */

    // area 1
    int sx = threadIdx.x,
        sy = threadIdx.y;
    int2 pix = block_orig + make_int2(sx,sy);
    if(lvl.contains(pix))
        fun(pix, sx, sy);

    // area 2
    pix.x += BW;
    sx += BW;
    if(lvl.contains(pix))
        fun(pix, sx, sy);

    // area 3
    pix.y += BH;
    sy += BH;
    if(lvl.contains(pix))
        fun(pix, sx, sy);

    // area 4
    pix.x -= BW;
    sx -= BW;
    if(lvl.contains(pix))
        fun(pix, sx, sy);

    // area 5
    sx = (threadIdx.y/4)*BW + threadIdx.x;
    sy = threadIdx.y%4 + BH*2;
    pix.x = block_orig.x+sx;
    pix.y = block_orig.y+sy;
    if(lvl.contains(pix))
        fun(pix, sx, sy);

    // area 6
    sx = threadIdx.x%4 + BW*2;
    sy = threadIdx.y*(BW/4) + threadIdx.x/4;
    pix.x = block_orig.x+sx;
    pix.y = block_orig.y+sy;
    if(lvl.contains(pix))
        fun(pix, sx, sy);

    // area 7
    sy += BH*2;
    pix.y += BH*2;
    if(lvl.contains(pix) && sy < BH*2+4)
        fun(pix, sx, sy);
}/*}}}*/

template <template<int,int> class F>
__device__ void process_shared_state(F<32,8> fun, const KernPyramidLevel &lvl,/*{{{*/
                                     const int2 &block_orig)
{
    const int BW = 32, BH = 8;

    int sx = threadIdx.x,
        sy = threadIdx.y;

    /*     BW      BW      4
       -----------------------
       |        |        |   | BH
       |   1    |   2    | 6 |
       |-----------------|---|
       |        |        |   | BH
       |   4    |   3    | 6 |
       |-----------------|---|
       |   5    |   5    | 6 | 4
       -----------------------
    */

    // area 1
    int2 pix = block_orig + make_int2(sx,sy);
    if(lvl.contains(pix))
        fun(pix, sx, sy);

    // area 2
    pix.x += BW;
    sx += BW;
    if(lvl.contains(pix))
        fun(pix, sx, sy);

    // area 3

    pix.y += BH;
    sy += BH;
    if(lvl.contains(pix))
        fun(pix, sx, sy);

    // area 4

    pix.x -= BW;
    sx -= BW;
    if(lvl.contains(pix))
        fun(pix, sx, sy);

    // area 5
    sx = (threadIdx.y/4)*BW + threadIdx.x;
    sy = threadIdx.y%4 + BH*2;
    pix.x = block_orig.x+sx;
    pix.y = block_orig.y+sy;
    if(lvl.contains(pix) && sx < BW*2+4 && sy < BH*2+4)
        fun(pix, sx, sy);

    // area 6
    sx = threadIdx.x%4 + BW*2;
    sy = threadIdx.y*8 + threadIdx.x/4;
    pix.x = block_orig.x+sx;
    pix.y = block_orig.y+sy;
    if(lvl.contains(pix) && sx < BW*2+4 && sy < BH*2+4)
        fun(pix, sx, sy);
}/*}}}*/

template <int BW, int BH>
class LoadSSIM/*{{{*/
{
public:
    __device__ LoadSSIM(const KernPyramidLevel &lvl, SSIMData<BW,BH> &ssim)
        : m_level(lvl), m_ssim(ssim) {}

    __device__ void operator()(const int2 &pix, int sx, int sy)
    {
        int idx = mem_index(m_level, pix);
        m_ssim.mean[sy][sx] = m_level.ssim.mean[idx];
        m_ssim.var[sy][sx] = m_level.ssim.var[idx];
        m_ssim.cross[sy][sx] = m_level.ssim.cross[idx];
        m_ssim.value[sy][sx] = m_level.ssim.value[idx];
    }

private:
    const KernPyramidLevel &m_level;
    SSIMData<BW,BH> &m_ssim;
};/*}}}*/

template <int BW, int BH>
class SaveSSIM/*{{{*/
{
public:
    __device__ SaveSSIM(KernPyramidLevel &lvl, const SSIMData<BW,BH> &ssim)
        : m_level(lvl), m_ssim(ssim) {}

    __device__ void operator()(const int2 &pix, int sx, int sy)
    {
        int idx = mem_index(m_level, pix);
        m_level.ssim.mean[idx] = m_ssim.mean[sy][sx];
        m_level.ssim.var[idx] = m_ssim.var[sy][sx];
        m_level.ssim.cross[idx] = m_ssim.cross[sy][sx];
        m_level.ssim.value[idx] = m_ssim.value[sy][sx];
    }

private:
    KernPyramidLevel &m_level;
    const SSIMData<BW,BH> &m_ssim;
};/*}}}*/

template <int BW, int BH>
class UpdateSSIM/*{{{*/
{
public:
    __device__ UpdateSSIM(const KernPyramidLevel &lvl, SSIMData<BW,BH> &ssim)
        : m_level(lvl), m_ssim(ssim) {}

    __device__ void operator()(const int2 &pix, int sx, int sy)
    {
        int idx = mem_index(m_level, pix);
        m_ssim.value[sy][sx] = ssim(m_ssim.mean[sy][sx],
                                    m_ssim.var[sy][sx],
                                    m_ssim.cross[sy][sx],
                                    m_level.ssim.counter[idx],
                                    c_params.ssim_clamp);
    }

private:
    const KernPyramidLevel &m_level;
    SSIMData<BW,BH> &m_ssim;
};/*}}}*/

__global__
//__launch_bounds__(OPT_BW*OPT_BH, OPT_NB)
void kernel_optimize_level(KernPyramidLevel lvl,/*{{{*/
                           int offx, int offy,
                           bool *out_improving)
{

    __shared__ SSIMData<OPT_BW,OPT_BH> ssim;

    {
        int2 block_orig = make_int2(blockIdx.x*(OPT_BW*2+SPACING)+offx-2,
                                    blockIdx.y*(OPT_BH*2+SPACING)+offy-2);

        if(threadIdx.x == 0 && threadIdx.y == 0)
            ssim.orig = block_orig;

        process_shared_state(LoadSSIM<OPT_BW,OPT_BH>(lvl, ssim), lvl, block_orig);
    }

    bool improving = false;

    __syncthreads();

    for(int i=0; i<2; ++i)
    {
        for(int j=0; j<2; ++j)
        {
            int2 p = ssim.orig + make_int2(threadIdx.x*2+j+2,
                                           threadIdx.y*2+i+2);

            float2 old_luma, v, grad;
            int impmask_idx = -1;
            bool ok = optimize_pixel(lvl, p, ssim, old_luma, v, grad, 
                                     impmask_idx);

            int2 offset = p%5;
            __syncthreads();

            if(ok)
            {
                commit_pixel_motion(lvl, p, v, old_luma, grad, ssim);

                improving = true;
                atomicOr(&lvl.improving_mask[impmask_idx], 
                         1 << (offset.x + offset.y*5));
            }
            else if(impmask_idx >= 0)
            {
                atomicAnd(&lvl.improving_mask[impmask_idx], 
                          ~(1 << (offset.x + offset.y*5)));
            }
            __syncthreads();

            process_shared_state(UpdateSSIM<OPT_BW,OPT_BH>(lvl, ssim), lvl, ssim.orig);

            __syncthreads();
        }
    }

    process_shared_state(SaveSSIM<OPT_BW,OPT_BH>(lvl, ssim), lvl, ssim.orig);

    if(improving)
        *out_improving = true;
}/*}}}*/

template <class T>
T *addressof(T &v)
{
    return reinterpret_cast<T*>(&const_cast<char &>(reinterpret_cast<const volatile char &>(v)));
}

bool Morph::optimize_level(int &curiter, int maxiter, int totaliter, 
                           PyramidLevel &lvl, int orig_width, int orig_height, 
                           int nlevel) const
{
    dim3 bdim(OPT_BW,OPT_BH),
         gdim((lvl.width+OPT_BW*2+SPACING-1)/(OPT_BW*2+SPACING),
              (lvl.height+OPT_BH*2+SPACING-1)/(OPT_BH*2+SPACING));

    rod::base_timer *timer = NULL;
    if(m_params.verbose)
        timer = &rod::timers.gpu_add("optimize",lvl.width*lvl.height,"P");

    KernPyramidLevel klvl(lvl);
    KernParameters kparams(m_params);

    rod::copy_to_symbol(c_params,kparams);

    bool *improving = NULL;
    hipHostAlloc(&improving, sizeof(bool), hipHostMallocMapped);
    rod::check_cuda_error("hipHostAlloc");
    assert(improving != NULL);

    bool *dimproving = NULL;
    hipHostGetDevicePointer(&dimproving, improving, 0);
    rod::check_cuda_error("hipHostGetDevicePointer");

    int iter = 0;

    rod::cpu_timer cb_send_image_timer(0,"",false);

    try
    {
        do
        {
            if(m_cb != NULL)
            {
                std::ostringstream ss;
                ss << "Optimizing level " << nlevel <<  " iteration #" << iter+1;

                rod::dimage<float2> halfway;

                clock_t now = clock();

                if(cb_send_image_timer.is_stopped() || 
                   cb_send_image_timer.elapsed() >= 0.2)
                {
                    cb_send_image_timer.start();

                    internal_vector_to_image(halfway, lvl.v, lvl,
                                             make_float2((float)orig_width/lvl.width,
                                                         (float)orig_height/lvl.height));

                    if(orig_width != lvl.width || orig_height != lvl.height)
                    {
                        rod::dimage<float2> temp(orig_width, orig_height);
                        upsample(&temp, &halfway, rod::INTERP_LINEAR);

                        swap(halfway,temp); // move temp to halfway
                    }
                }

                if(!m_cb(ss.str(), ++curiter, totaliter, 
                       halfway.empty() ? NULL : addressof(halfway), NULL, 
                       m_cbdata))
                {
                    if(timer)
                        timer->stop();

                    return false;
                }
            }

            *improving = false;

            kernel_optimize_level<<<gdim,bdim>>>(klvl, 0,0, dimproving);
            kernel_optimize_level<<<gdim,bdim>>>(klvl, OPT_BW*2, 0, dimproving);
            kernel_optimize_level<<<gdim,bdim>>>(klvl, 0, OPT_BH*2, dimproving);
            kernel_optimize_level<<<gdim,bdim>>>(klvl, OPT_BW*2, OPT_BH*2, dimproving);

            hipDeviceSynchronize();

            ++iter;
        }
        while(*improving && iter <= maxiter);

        curiter += maxiter-iter;

        hipHostFree(improving);
    }
    catch(...)
    {
        if(timer)
            timer->stop();

        cb_send_image_timer.stop();


        hipHostFree(improving);
        throw;
    }

    cb_send_image_timer.stop();

    if(timer)
        timer->stop();

    if(m_params.verbose)
        std::cout << iter << " iterations, grid " << gdim.x << 'x' << gdim.y << std::endl;

    return true;
}
